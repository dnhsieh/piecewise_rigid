#include <hipblas.h>
#include <hipsolver.h>
#include "struct.h"
#include "constants.h"

void   group2Node(double *, double *, int *, int, int, int);
double computeControlCost(double *, double *, fcndata &);
void   computeKernel(double *, double *, fcndata &);
void   computeKernel(double *, double *, double *, fcndata &);
void   addEpsIdentity(double *, double, int);
void   cholesky(double *, fcndata &);
void   computeRigidVelocity(double *, double *, double *, fcndata &);
void   computeRigidDeformation(double *, double *, fcndata &);

void   duRigidVelocity(double *, double *, double *, fcndata &);
void   dqKernel(double *, double *, double *, double *, fcndata &);
void   dqRigidVelocity(double *, double *, double *, double *, fcndata &);
void   dqRigidDeformation(double *, double *, double *, fcndata &);
void   duControl(double *, double *, double *, fcndata &);
void   node2Group(double *, double *, int *, int, int, int);

void   vectorSum(double *, double, double *, double, double *, int);
void   vectorSum(double *, double, double *, double, double *, double, double *, int);
void   vectorSum(double *, double, double *, double, double *, double, double *, double, double *, int);
void   vectorScale(double *, double, double *, int);

void objgrd(double *h_objPtr, double *d_grdGrpStk, double *d_ctlGrpStk, fcndata &fcnObj)
{
	int    rgdGrpNum = fcnObj.prm.rgdGrpNum;
	int    rgdNdeNum = fcnObj.prm.rgdNdeNum;
	int      timeNum = fcnObj.prm.timeNum;
	double   timeStp = fcnObj.prm.timeStp;

	hipMemset(fcnObj.d_grdNdeStk, 0, sizeof(double) * rgdNdeNum * RGDDOF * (timeNum - 1));

	group2Node(fcnObj.d_ctlNdeStk, d_ctlGrpStk, fcnObj.prm.d_grpNdeVec,
	           rgdNdeNum, rgdGrpNum, timeNum);

	hipMemset(fcnObj.d_ctlCumStk, 0, sizeof(double) * rgdNdeNum * DIMNUM);
	hipMemcpy(fcnObj.d_ctlCumStk + rgdNdeNum * DIMNUM, fcnObj.prm.d_cenIniMat,
	           sizeof(double) * rgdNdeNum * DIMNUM, hipMemcpyDeviceToDevice);

	computeRigidDeformation(fcnObj.d_rgdNdeStk, fcnObj.d_ctlCumStk, fcnObj);	

	*h_objPtr = computeControlCost(d_ctlGrpStk, fcnObj.prm.d_wgtGrpVec, fcnObj);

	for ( int timeIdx = 0; timeIdx < timeNum - 1; ++timeIdx )
	{
		fcnObj.d_ctlNdeMat    = fcnObj.d_ctlNdeStk +  timeIdx      * rgdNdeNum * RGDDOF;
		fcnObj.d_ctlCumNowMat = fcnObj.d_ctlCumStk +  timeIdx      * rgdNdeNum * RGDDOF;
		fcnObj.d_ctlCumNxtMat = fcnObj.d_ctlCumStk + (timeIdx + 1) * rgdNdeNum * RGDDOF;
		fcnObj.d_rgdNdeNowMat = fcnObj.d_rgdNdeStk +  timeIdx      * rgdNdeNum * DIMNUM;
		fcnObj.d_rgdNdeNxtMat = fcnObj.d_rgdNdeStk + (timeIdx + 1) * rgdNdeNum * DIMNUM;
		fcnObj.d_rgdVlcMat    = fcnObj.d_rgdVlcStk +  timeIdx      * rgdNdeNum * DIMNUM;
		fcnObj.d_rgdAlpMat    = fcnObj.d_rgdAlpStk +  timeIdx      * rgdNdeNum * DIMNUM;

		computeRigidVelocity(fcnObj.d_rgdVlcMat, fcnObj.d_ctlNdeMat, fcnObj.d_ctlCumNowMat, fcnObj);

		computeKernel(fcnObj.d_rgdKnlMat, fcnObj.d_rgdNdeNowMat, fcnObj); 
		addEpsIdentity(fcnObj.d_rgdKnlMat, fcnObj.prm.knlEps, rgdNdeNum);

		hipMemcpy(fcnObj.d_rgdKnLMat, fcnObj.d_rgdKnlMat,
		           sizeof(double) * rgdNdeNum * rgdNdeNum, hipMemcpyDeviceToDevice);
		cholesky(fcnObj.d_rgdKnLMat, fcnObj);

		hipMemcpy(fcnObj.d_rgdAlpMat, fcnObj.d_rgdVlcMat,
		           sizeof(double) * rgdNdeNum * DIMNUM, hipMemcpyDeviceToDevice);
		hipsolverDnDpotrs(fcnObj.solvHdl, HIPBLAS_FILL_MODE_LOWER, rgdNdeNum, DIMNUM,
		                 fcnObj.d_rgdKnLMat, rgdNdeNum,
		                 fcnObj.d_rgdAlpMat, rgdNdeNum, fcnObj.d_status);

		double h_ldmVal;
		hipblasDdot(fcnObj.blasHdl, rgdNdeNum * DIMNUM,
		           fcnObj.d_rgdAlpMat, 1, fcnObj.d_rgdVlcMat, 1, &h_ldmVal);
		*h_objPtr += 0.5 * h_ldmVal;

		vectorSum(fcnObj.d_ctlCumNxtMat,
		          1.0, fcnObj.d_ctlCumNowMat, timeStp, fcnObj.d_ctlNdeMat, rgdNdeNum * RGDDOF);
	
		computeRigidDeformation(fcnObj.d_rgdNdeNxtMat, fcnObj.d_ctlCumNxtMat, fcnObj);
	}

	double *d_rgdNdeEndMat = fcnObj.d_rgdNdeStk + (timeNum - 1) * rgdNdeNum * DIMNUM;

	vectorSum(fcnObj.d_difMat, 1.0, d_rgdNdeEndMat, -1.0, fcnObj.tgt.d_tgtNdeMat, rgdNdeNum * DIMNUM);

	double h_l2SquVal;
	hipblasDdot(fcnObj.blasHdl, rgdNdeNum * DIMNUM, fcnObj.d_difMat, 1, fcnObj.d_difMat, 1, &h_l2SquVal);

	*h_objPtr = timeStp * (*h_objPtr) + fcnObj.prm.tgtWgt * 0.5 * h_l2SquVal;

	double *d_ctlCumEndMat = fcnObj.d_ctlCumStk + (timeNum - 1) * rgdNdeNum * RGDDOF;
	dqRigidDeformation(fcnObj.d_pMat, d_ctlCumEndMat, fcnObj.d_difMat, fcnObj);
	vectorScale(fcnObj.d_pMat, -fcnObj.prm.tgtWgt, fcnObj.d_pMat, rgdNdeNum * RGDDOF);

	for ( int timeIdx = timeNum - 2; timeIdx >= 0; --timeIdx )
	{
		fcnObj.d_grdNdeMat    = fcnObj.d_grdNdeStk + timeIdx * rgdNdeNum * RGDDOF;

		fcnObj.d_ctlNdeMat    = fcnObj.d_ctlNdeStk + timeIdx * rgdNdeNum * RGDDOF;
		fcnObj.d_ctlCumNowMat = fcnObj.d_ctlCumStk + timeIdx * rgdNdeNum * RGDDOF;
		fcnObj.d_rgdNdeNowMat = fcnObj.d_rgdNdeStk + timeIdx * rgdNdeNum * DIMNUM;
		fcnObj.d_rgdAlpMat    = fcnObj.d_rgdAlpStk + timeIdx * rgdNdeNum * DIMNUM;

		duRigidVelocity(fcnObj.d_duRVMat, fcnObj.d_ctlCumNowMat, fcnObj.d_rgdAlpMat, fcnObj);
		vectorSum(fcnObj.d_grdNdeMat, 1.0, fcnObj.d_duRVMat, -1.0, fcnObj.d_pMat, rgdNdeNum * RGDDOF);

		dqKernel(fcnObj.d_dqKMat, fcnObj.d_rgdNdeNowMat, fcnObj.d_rgdAlpMat, fcnObj.d_rgdAlpMat, fcnObj);
		vectorScale(fcnObj.d_dqKMat, -0.5, fcnObj.d_dqKMat, rgdNdeNum * DIMNUM);

		dqRigidDeformation(fcnObj.d_dqRDMat, fcnObj.d_ctlCumNowMat, fcnObj.d_dqKMat, fcnObj);
		dqRigidVelocity(fcnObj.d_dqRVMat, fcnObj.d_ctlNdeMat, fcnObj.d_ctlCumNowMat,
		                fcnObj.d_rgdAlpMat, fcnObj);

		vectorSum(fcnObj.d_pDotMat, 1.0, fcnObj.d_dqRDMat, 1.0,     fcnObj.d_dqRVMat, rgdNdeNum * RGDDOF);
		vectorSum(fcnObj.d_pMat,    1.0, fcnObj.d_pMat,   -timeStp, fcnObj.d_pDotMat, rgdNdeNum * RGDDOF);
	}

	node2Group(d_grdGrpStk, fcnObj.d_grdNdeStk, fcnObj.prm.d_grpIfoMat,
	           rgdNdeNum, rgdGrpNum, timeNum);
	duControl(d_grdGrpStk, d_ctlGrpStk, fcnObj.prm.d_wgtGrpVec, fcnObj);
	vectorScale(d_grdGrpStk, timeStp, d_grdGrpStk, rgdGrpNum * RGDDOF * (timeNum - 1));

	return;
}
