#include <cstdio>
#include <cstdlib>
#include <cfloat>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "struct.h"
#include "constants.h"

void assignObjfcnStructMemory(long long &, fcndata &, double *);
void objfcn(double *, double *, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	fcndata fcnObj = {0};

	mxGPUArray const *ctlGrpStk;
	mxGPUArray const *cenIniMat, *difIniMat, *grpNdeVec, *wgtGrpVec;
	mxGPUArray const *vfdElmVtxMat, *tgtCenPosMat, *tgtUniDirMat, *tgtElmVolVec;

	ctlGrpStk              =  mxGPUCreateFromMxArray(prhs[ 0]);
	cenIniMat              =  mxGPUCreateFromMxArray(prhs[ 1]);
	difIniMat              =  mxGPUCreateFromMxArray(prhs[ 2]);
	grpNdeVec              =  mxGPUCreateFromMxArray(prhs[ 3]);
	wgtGrpVec              =  mxGPUCreateFromMxArray(prhs[ 4]);
	vfdElmVtxMat           =  mxGPUCreateFromMxArray(prhs[ 5]);
	tgtCenPosMat           =  mxGPUCreateFromMxArray(prhs[ 6]);
	tgtUniDirMat           =  mxGPUCreateFromMxArray(prhs[ 7]);
	tgtElmVolVec           =  mxGPUCreateFromMxArray(prhs[ 8]);
	fcnObj.vfd.cenKnlType  =             mxGetScalar(prhs[ 9]);
	fcnObj.vfd.cenKnlWidth =             mxGetScalar(prhs[10]);
	fcnObj.vfd.dirKnlType  =             mxGetScalar(prhs[11]);
	fcnObj.vfd.dirKnlWidth =             mxGetScalar(prhs[12]);
	fcnObj.prm.knlOrder    =             mxGetScalar(prhs[13]);
	fcnObj.prm.knlWidth    =             mxGetScalar(prhs[14]);
	fcnObj.prm.knlEps      =             mxGetScalar(prhs[15]);
	fcnObj.prm.timeStp     =             mxGetScalar(prhs[16]);
	fcnObj.prm.timeNum     =             mxGetScalar(prhs[17]);
	fcnObj.prm.tgtWgt      =             mxGetScalar(prhs[18]);

	// ---

	double *d_ctlGrpStk = (double *) mxGPUGetDataReadOnly(ctlGrpStk);

	fcnObj.prm.d_cenIniMat    = (double *) mxGPUGetDataReadOnly(cenIniMat);
	fcnObj.prm.d_difIniMat    = (double *) mxGPUGetDataReadOnly(difIniMat);
	fcnObj.prm.d_grpNdeVec    = (int    *) mxGPUGetDataReadOnly(grpNdeVec);
	fcnObj.prm.d_wgtGrpVec    = (double *) mxGPUGetDataReadOnly(wgtGrpVec);
	fcnObj.elm.d_vfdElmVtxMat = (int    *) mxGPUGetDataReadOnly(vfdElmVtxMat);
	fcnObj.tgt.d_cenPosMat    = (double *) mxGPUGetDataReadOnly(tgtCenPosMat);
	fcnObj.tgt.d_uniDirMat    = (double *) mxGPUGetDataReadOnly(tgtUniDirMat);
	fcnObj.tgt.d_elmVolVec    = (double *) mxGPUGetDataReadOnly(tgtElmVolVec);

	mwSize const *vfdElmDims = mxGPUGetDimensions(vfdElmVtxMat);
	mwSize const *tgtElmDims = mxGPUGetDimensions(tgtCenPosMat);

	fcnObj.prm.rgdGrpNum = mxGPUGetNumberOfElements(wgtGrpVec);
	fcnObj.prm.rgdNdeNum = mxGPUGetNumberOfElements(grpNdeVec);
	fcnObj.prm.vfdNdeNum = fcnObj.prm.rgdNdeNum;
	fcnObj.prm.vfdElmNum = vfdElmDims[0];
	fcnObj.tgt.tgtElmNum = tgtElmDims[0];

	// ---

	int rgdGrpNum = fcnObj.prm.rgdGrpNum;
	int rgdNdeNum = fcnObj.prm.rgdNdeNum;
	int vfdElmNum = fcnObj.prm.vfdElmNum;
	int timeNum   = fcnObj.prm.timeNum;

	long long gpuAloDblMemCnt =  rgdGrpNum * (timeNum - 1)
	                           + rgdNdeNum * (  rgdNdeNum * 2 + DIMNUM * timeNum + DIMNUM * (timeNum - 1) * 2
	                                          + RGDDOF * (timeNum - 1) + RGDDOF * timeNum)
	                           + vfdElmNum * (DIMNUM * 2 + 2) + fcnObj.tgt.tgtElmNum 
	                           + SUMBLKDIM;

	double *gpuDblSpace;
	hipError_t error = hipMalloc((void **) &gpuDblSpace, sizeof(double) * gpuAloDblMemCnt);
	if ( error != hipSuccess )
		mexErrMsgIdAndTxt("objfcn2D:hipMalloc", "Fail to allocate device memory.");

	hipMalloc((void **) &(fcnObj.d_status), sizeof(int));

	long long gpuAsgDblMemCnt;
	assignObjfcnStructMemory(gpuAsgDblMemCnt, fcnObj, gpuDblSpace);
	if ( gpuAsgDblMemCnt != gpuAloDblMemCnt )
	{
		mexErrMsgIdAndTxt("objfcn2D:memAssign", 
		                  "Assigned device double memory (%lld) mismatches the allocated memory (%lld).", 
		                  gpuAsgDblMemCnt, gpuAloDblMemCnt);
	}

	// ---

	hipblasCreate(&(fcnObj.blasHdl));

	hipsolverDnCreate(&(fcnObj.solvHdl));
	hipsolverDnDpotrf_bufferSize(fcnObj.solvHdl, HIPBLAS_FILL_MODE_LOWER,
	                            fcnObj.prm.rgdNdeNum, fcnObj.d_rgdKnlMat,
	                            fcnObj.prm.rgdNdeNum, &(fcnObj.h_Lwork));

	hipMalloc((void **) &(fcnObj.d_workspace), sizeof(double) * fcnObj.h_Lwork);

	// ---

	double h_objVal;
	objfcn(&h_objVal, d_ctlGrpStk, fcnObj);

	plhs[0] = mxCreateDoubleScalar(h_objVal);

	// ---
	//

	mxGPUDestroyGPUArray(ctlGrpStk);
	mxGPUDestroyGPUArray(cenIniMat);
	mxGPUDestroyGPUArray(difIniMat);
	mxGPUDestroyGPUArray(grpNdeVec);
	mxGPUDestroyGPUArray(wgtGrpVec);
	mxGPUDestroyGPUArray(vfdElmVtxMat);
	mxGPUDestroyGPUArray(tgtCenPosMat);
	mxGPUDestroyGPUArray(tgtUniDirMat);
	mxGPUDestroyGPUArray(tgtElmVolVec);

	mxFree((void *) vfdElmDims);
	mxFree((void *) tgtElmDims);

	hipFree(gpuDblSpace);
	hipFree(fcnObj.d_status);
	hipFree(fcnObj.d_workspace);

	hipblasDestroy(fcnObj.blasHdl);
	hipsolverDnDestroy(fcnObj.solvHdl);

	return;
}

