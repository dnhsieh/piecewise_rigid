#include <cstdio>
#include <cstdlib>
#include <cfloat>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "struct.h"
#include "constants.h"

void assignOptimizationStructMemory(long long &, optdata &, double *);
void assignObjgrdStructMemory(long long &, fcndata &, double *);
int  BFGS(double *, double *, double *, optdata &, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	optdata optObj = {0};
	fcndata fcnObj = {0};

	mxGPUArray const *optIniStk;
	mxGPUArray const *cenIniMat, *difIniMat, *grpNdeVec, *grpIfoMat, *wgtGrpVec;
	mxGPUArray const *vfdElmVtxMat, *vfdElmIfoMat;
	mxGPUArray const *tgtCenPosMat, *tgtUniDirMat, *tgtElmVolVec;

	mxGPUArray       *posNowStk, *grdNowStk;

	optIniStk              =  mxGPUCreateFromMxArray(prhs[ 0]);
	optObj.itrMax          =             mxGetScalar(prhs[ 1]);
	optObj.tolVal          =             mxGetScalar(prhs[ 2]);
	optObj.wolfe1          =             mxGetScalar(prhs[ 3]);
	optObj.wolfe2          =             mxGetScalar(prhs[ 4]);
	optObj.vbsFlg          =             mxGetScalar(prhs[ 5]);
	cenIniMat              =  mxGPUCreateFromMxArray(prhs[ 6]);
	difIniMat              =  mxGPUCreateFromMxArray(prhs[ 7]);
	grpNdeVec              =  mxGPUCreateFromMxArray(prhs[ 8]);
	grpIfoMat              =  mxGPUCreateFromMxArray(prhs[ 9]);
	wgtGrpVec              =  mxGPUCreateFromMxArray(prhs[10]);
	vfdElmVtxMat           =  mxGPUCreateFromMxArray(prhs[11]);
	vfdElmIfoMat           =  mxGPUCreateFromMxArray(prhs[12]);
	tgtCenPosMat           =  mxGPUCreateFromMxArray(prhs[13]);
	tgtUniDirMat           =  mxGPUCreateFromMxArray(prhs[14]);
	tgtElmVolVec           =  mxGPUCreateFromMxArray(prhs[15]);
	fcnObj.vfd.cenKnlType  =             mxGetScalar(prhs[16]);
	fcnObj.vfd.cenKnlWidth =             mxGetScalar(prhs[17]);
	fcnObj.vfd.dirKnlType  =             mxGetScalar(prhs[18]);
	fcnObj.vfd.dirKnlWidth =             mxGetScalar(prhs[19]);
	fcnObj.prm.knlOrder    =             mxGetScalar(prhs[20]);
	fcnObj.prm.knlWidth    =             mxGetScalar(prhs[21]);
	fcnObj.prm.knlEps      =             mxGetScalar(prhs[22]);
	fcnObj.prm.timeStp     =             mxGetScalar(prhs[23]);
	fcnObj.prm.timeNum     =             mxGetScalar(prhs[24]);
	fcnObj.prm.tgtWgt      =             mxGetScalar(prhs[25]);

	fcnObj.prm.rgdGrpNum = mxGPUGetNumberOfElements(wgtGrpVec);

	mwSize const ndim = 3;
	mwSize const dims[3] = {(mwSize) fcnObj.prm.rgdGrpNum, (mwSize) RGDDOF, (mwSize) (fcnObj.prm.timeNum - 1)};
	posNowStk = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	grdNowStk = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	
	// ---

	double *d_optIniStk = (double *) mxGPUGetDataReadOnly(optIniStk);

	fcnObj.prm.d_cenIniMat    = (double *) mxGPUGetDataReadOnly(cenIniMat);
	fcnObj.prm.d_difIniMat    = (double *) mxGPUGetDataReadOnly(difIniMat);
	fcnObj.prm.d_grpNdeVec    = (int    *) mxGPUGetDataReadOnly(grpNdeVec);
	fcnObj.prm.d_grpIfoMat    = (int    *) mxGPUGetDataReadOnly(grpIfoMat);
	fcnObj.prm.d_wgtGrpVec    = (double *) mxGPUGetDataReadOnly(wgtGrpVec);
	fcnObj.elm.d_vfdElmVtxMat = (int    *) mxGPUGetDataReadOnly(vfdElmVtxMat);
	fcnObj.elm.d_vfdElmIfoMat = (int    *) mxGPUGetDataReadOnly(vfdElmIfoMat);
	fcnObj.tgt.d_cenPosMat    = (double *) mxGPUGetDataReadOnly(tgtCenPosMat);
	fcnObj.tgt.d_uniDirMat    = (double *) mxGPUGetDataReadOnly(tgtUniDirMat);
	fcnObj.tgt.d_elmVolVec    = (double *) mxGPUGetDataReadOnly(tgtElmVolVec);

	double  h_objVal;
	double *d_posNowStk = (double *) mxGPUGetData(posNowStk);
	double *d_grdNowStk = (double *) mxGPUGetData(grdNowStk);

	mwSize const *vfdElmDims = mxGPUGetDimensions(vfdElmVtxMat);
	mwSize const *tgtElmDims = mxGPUGetDimensions(tgtCenPosMat);

	fcnObj.prm.rgdNdeNum = mxGPUGetNumberOfElements(grpNdeVec);
	fcnObj.prm.vfdNdeNum = fcnObj.prm.rgdNdeNum;
	fcnObj.prm.vfdElmNum = vfdElmDims[0];
	fcnObj.tgt.tgtElmNum = tgtElmDims[0];

	int optVarNum = mxGPUGetNumberOfElements(optIniStk);
	optObj.varNum     = optVarNum;
	fcnObj.prm.varNum = optVarNum;

	// ---

	int rgdGrpNum = fcnObj.prm.rgdGrpNum;
	int rgdNdeNum = fcnObj.prm.rgdNdeNum;
	int vfdElmNum = fcnObj.prm.vfdElmNum;
	int timeNum   = fcnObj.prm.timeNum;

	long long optAloDblMemCnt =  optVarNum * (optVarNum + 6);
	long long gpuAloDblMemCnt =  rgdGrpNum * (timeNum - 1)
	                           + rgdNdeNum * (  rgdNdeNum * 2 + DIMNUM * timeNum + DIMNUM * (timeNum - 1) * 2
	                                          + RGDDOF * (timeNum - 1) + RGDDOF * timeNum)
	                           + vfdElmNum * (DIMNUM * 2 + 2) + fcnObj.tgt.tgtElmNum 
	                           + rgdNdeNum * (DIMNUM * 2 + RGDDOF * (timeNum - 1) + RGDDOF * 5)
	                           + vfdElmNum * DIMNUM * 2
	                           + SUMBLKDIM;

	double *optDblSpace;
	hipError_t error = hipMalloc((void **) &optDblSpace, sizeof(double) * optAloDblMemCnt);
	if ( error != hipSuccess )
		mexErrMsgIdAndTxt("rigidMatching2D:hipMalloc", "Fail to allocate device memory.");

	double *gpuDblSpace;
	error = hipMalloc((void **) &gpuDblSpace, sizeof(double) * gpuAloDblMemCnt);
	if ( error != hipSuccess )
		mexErrMsgIdAndTxt("rigidMatching2D:hipMalloc", "Fail to allocate device memory.");

	hipMalloc((void **) &(fcnObj.d_status), sizeof(int));

	long long optAsgDblMemCnt;
	assignOptimizationStructMemory(optAsgDblMemCnt, optObj, optDblSpace);
	if ( optAsgDblMemCnt != optAloDblMemCnt )
	{
		mexErrMsgIdAndTxt("rigidMatching2D:memAssign", 
		                  "Assigned device double memory (%lld) mismatches the allocated memory (%lld).", 
		                  optAsgDblMemCnt, optAloDblMemCnt);
	}

	long long gpuAsgDblMemCnt;
	assignObjgrdStructMemory(gpuAsgDblMemCnt, fcnObj, gpuDblSpace);
	if ( gpuAsgDblMemCnt != gpuAloDblMemCnt )
	{
		mexErrMsgIdAndTxt("rigidMatching2D:memAssign", 
		                  "Assigned device double memory (%lld) mismatches the allocated memory (%lld).", 
		                  gpuAsgDblMemCnt, gpuAloDblMemCnt);
	}

	// ---

	hipblasCreate(&(fcnObj.blasHdl));

	hipsolverDnCreate(&(fcnObj.solvHdl));
	hipsolverDnDpotrf_bufferSize(fcnObj.solvHdl, HIPBLAS_FILL_MODE_LOWER,
	                            fcnObj.prm.rgdNdeNum, fcnObj.d_rgdKnlMat,
	                            fcnObj.prm.rgdNdeNum, &(fcnObj.h_Lwork));

	hipMalloc((void **) &(fcnObj.d_workspace), sizeof(double) * fcnObj.h_Lwork);

	// ---

	hipMemcpy(d_posNowStk, d_optIniStk, sizeof(double) * optVarNum, hipMemcpyDeviceToDevice);
	BFGS(&h_objVal, d_grdNowStk, d_posNowStk, optObj, fcnObj);

	plhs[0] =    mxCreateDoubleScalar(h_objVal );
	plhs[1] = mxGPUCreateMxArrayOnGPU(posNowStk);
	plhs[2] = mxGPUCreateMxArrayOnGPU(grdNowStk);

	// ---
	//

	mxGPUDestroyGPUArray(optIniStk);
	mxGPUDestroyGPUArray(cenIniMat);
	mxGPUDestroyGPUArray(difIniMat);
	mxGPUDestroyGPUArray(grpNdeVec);
	mxGPUDestroyGPUArray(grpIfoMat);
	mxGPUDestroyGPUArray(wgtGrpVec);
	mxGPUDestroyGPUArray(vfdElmVtxMat);
	mxGPUDestroyGPUArray(vfdElmIfoMat);
	mxGPUDestroyGPUArray(tgtCenPosMat);
	mxGPUDestroyGPUArray(tgtUniDirMat);
	mxGPUDestroyGPUArray(tgtElmVolVec);
	mxGPUDestroyGPUArray(posNowStk);
	mxGPUDestroyGPUArray(grdNowStk);

	mxFree((void *) vfdElmDims);
	mxFree((void *) tgtElmDims);

	hipFree(optDblSpace);
	hipFree(gpuDblSpace);
	hipFree(fcnObj.d_status);
	hipFree(fcnObj.d_workspace);

	hipblasDestroy(fcnObj.blasHdl);
	hipsolverDnDestroy(fcnObj.solvHdl);

	return;
}

