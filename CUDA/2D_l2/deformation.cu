#include <cstdio>
#include <cstdlib>
#include <cfloat>
#include "mex.h"
#include "gpu/mxGPUArray.h"
#include <hipblas.h>
#include <hipsolver.h>
#include "struct.h"
#include "constants.h"

void assignObjfcnStructMemory(long long &, fcndata &, double *);
void objfcn(double *, double *, fcndata &);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, mxArray const *prhs[])
{
	mxInitGPU();

	fcndata fcnObj = {0};

	mxGPUArray const *ctlGrpStk;
	mxGPUArray const *cenIniMat, *difIniMat, *grpNdeVec, *wgtGrpVec, *tgtNdeMat;

	mxGPUArray       *rgdNdeStk;

	ctlGrpStk              =  mxGPUCreateFromMxArray(prhs[ 0]);
	cenIniMat              =  mxGPUCreateFromMxArray(prhs[ 1]);
	difIniMat              =  mxGPUCreateFromMxArray(prhs[ 2]);
	grpNdeVec              =  mxGPUCreateFromMxArray(prhs[ 3]);
	wgtGrpVec              =  mxGPUCreateFromMxArray(prhs[ 4]);
	tgtNdeMat              =  mxGPUCreateFromMxArray(prhs[ 5]);
	fcnObj.prm.knlOrder    =             mxGetScalar(prhs[ 6]);
	fcnObj.prm.knlWidth    =             mxGetScalar(prhs[ 7]);
	fcnObj.prm.knlEps      =             mxGetScalar(prhs[ 8]);
	fcnObj.prm.timeStp     =             mxGetScalar(prhs[ 9]);
	fcnObj.prm.timeNum     =             mxGetScalar(prhs[10]);
	fcnObj.prm.tgtWgt      =             mxGetScalar(prhs[11]);

	fcnObj.prm.rgdNdeNum = mxGPUGetNumberOfElements(grpNdeVec);

	mwSize const ndim = 3;
	mwSize const dims[3] = {(mwSize) fcnObj.prm.rgdNdeNum, (mwSize) DIMNUM, (mwSize) fcnObj.prm.timeNum};
	rgdNdeStk = mxGPUCreateGPUArray(ndim, dims, mxDOUBLE_CLASS, mxREAL, MX_GPU_DO_NOT_INITIALIZE);
	
	// ---

	double *d_ctlGrpStk = (double *) mxGPUGetDataReadOnly(ctlGrpStk);

	fcnObj.prm.d_cenIniMat    = (double *) mxGPUGetDataReadOnly(cenIniMat);
	fcnObj.prm.d_difIniMat    = (double *) mxGPUGetDataReadOnly(difIniMat);
	fcnObj.prm.d_grpNdeVec    = (int    *) mxGPUGetDataReadOnly(grpNdeVec);
	fcnObj.prm.d_wgtGrpVec    = (double *) mxGPUGetDataReadOnly(wgtGrpVec);
	fcnObj.tgt.d_tgtNdeMat    = (double *) mxGPUGetDataReadOnly(tgtNdeMat);

	double *d_rgdNdeStk = (double *) mxGPUGetData(rgdNdeStk);

	fcnObj.prm.rgdGrpNum = mxGPUGetNumberOfElements(wgtGrpVec);

	// ---

	int rgdGrpNum = fcnObj.prm.rgdGrpNum;
	int rgdNdeNum = fcnObj.prm.rgdNdeNum;
	int timeNum   = fcnObj.prm.timeNum;

	long long gpuAloDblMemCnt =  rgdGrpNum * (timeNum - 1)
	                           + rgdNdeNum * (  rgdNdeNum * 2 + DIMNUM + DIMNUM * timeNum + DIMNUM * (timeNum - 1) * 2
	                                          + RGDDOF * (timeNum - 1) + RGDDOF * timeNum)
	                           + SUMBLKDIM;

	double *gpuDblSpace;
	hipError_t error = hipMalloc((void **) &gpuDblSpace, sizeof(double) * gpuAloDblMemCnt);
	if ( error != hipSuccess )
		mexErrMsgIdAndTxt("deformation2D:hipMalloc", "Fail to allocate device memory.");

	hipMalloc((void **) &(fcnObj.d_status), sizeof(int));

	long long gpuAsgDblMemCnt;
	assignObjfcnStructMemory(gpuAsgDblMemCnt, fcnObj, gpuDblSpace);
	if ( gpuAsgDblMemCnt != gpuAloDblMemCnt )
	{
		mexErrMsgIdAndTxt("deformation2D:memAssign", 
		                  "Assigned device double memory (%lld) mismatches the allocated memory (%lld).", 
		                  gpuAsgDblMemCnt, gpuAloDblMemCnt);
	}

	// ---

	hipblasCreate(&(fcnObj.blasHdl));

	hipsolverDnCreate(&(fcnObj.solvHdl));
	hipsolverDnDpotrf_bufferSize(fcnObj.solvHdl, HIPBLAS_FILL_MODE_LOWER,
	                            fcnObj.prm.rgdNdeNum, fcnObj.d_rgdKnlMat,
	                            fcnObj.prm.rgdNdeNum, &(fcnObj.h_Lwork));

	hipMalloc((void **) &(fcnObj.d_workspace), sizeof(double) * fcnObj.h_Lwork);

	// ---

	double h_objVal;
	objfcn(&h_objVal, d_ctlGrpStk, fcnObj);

	hipMemcpy(d_rgdNdeStk, fcnObj.d_rgdNdeStk,
	           sizeof(double) * rgdNdeNum * DIMNUM * timeNum, hipMemcpyDeviceToDevice);
	plhs[0] = mxGPUCreateMxArrayOnGPU(rgdNdeStk);

	// ---
	//

	mxGPUDestroyGPUArray(ctlGrpStk);
	mxGPUDestroyGPUArray(cenIniMat);
	mxGPUDestroyGPUArray(difIniMat);
	mxGPUDestroyGPUArray(grpNdeVec);
	mxGPUDestroyGPUArray(wgtGrpVec);
	mxGPUDestroyGPUArray(tgtNdeMat);
	mxGPUDestroyGPUArray(rgdNdeStk);

	hipFree(gpuDblSpace);
	hipFree(fcnObj.d_status);
	hipFree(fcnObj.d_workspace);

	hipblasDestroy(fcnObj.blasHdl);
	hipsolverDnDestroy(fcnObj.solvHdl);

	return;
}

